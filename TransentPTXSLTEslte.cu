
#include <hip/hip_runtime.h>
__global__ void transent_1(double *te_result, double *slte_result,
  double *all_series, const int *positions, const double *lengths, 
  const int series_count,
 const int y_delay,
 const int duration, const double *rule)

 {                               /// TERMWISE

   int cell = blockDim.x*blockIdx.x + threadIdx.x;

   if (cell >= series_count*series_count) {
       return;
   }

  /* Constants */
  const unsigned int x_order = 1, y_order = 1,
               num_series = 3,
               num_counts = 8,
               num_x = 4,
               num_y = 2;

  /* Locals */
  int counts[8];
  unsigned long code;
  long k, l, idx, c1, c2;
  double te_final, prob_1, prob_2, prob_3;

  double *ord_iter[3]; ////
  double *ord_end[3];  ////

  int ord_times[3];
  int ord_shift[3];

  const unsigned int window = y_order + y_delay;
  const int end_time = duration - window + 1;
  int cur_time, next_time;

  /* Calculate TE */
  // double *array_ptr;
  double *i_series, *j_series; ////
  int i_size, j_size;
  int i, j;


  /* MATLAB is column major */
  i = cell/series_count;
  j = cell % series_count;

      /* Extract series */
      i_size = lengths[i];
      i_series = &all_series[positions[i]];

      j_size = lengths[j];
    j_series = &all_series[positions[j]];

      if ((i_size == 0) || (j_size == 0)) {
        te_result[(i * series_count) + j] = 0;

		//continue;
    return;
      }

      /* Order is x^(k+1), y^(l) */
      idx = 0;

      /* x^(k+1) */
      for (k = 0; k < (x_order + 1); ++k) {
        ord_iter[idx] = i_series;
        ord_end[idx] = i_series + i_size;
        ord_shift[idx] = (window - 1) - k;

        while ((int)*(ord_iter[idx]) < ord_shift[idx] + 1) {
          ++(ord_iter[idx]);
        }

        ord_times[idx] = (int)*(ord_iter[idx]) - ord_shift[idx];
        ++idx;
      }

      /* y^(l) */
      for (k = 0; k < y_order; ++k) {
        ord_iter[idx] = j_series;
        ord_end[idx] = j_series + j_size;
        ord_shift[idx] = -k;
        ord_times[idx] = (int)*(ord_iter[idx]) - ord_shift[idx];
        ++idx;
      }

      /* Count spikes */
      memset(counts, 0, sizeof(int) * num_counts);

      /* Get minimum next time bin */
      cur_time = ord_times[0];
      for (k = 1; k < num_series; ++k) {
        if (ord_times[k] < cur_time) {
          cur_time = ord_times[k];
        }
      }

      while (cur_time <= end_time) {

        code = 0;
        next_time = end_time + 1;

        /* Calculate hash code for this time bin */
        for (k = 0; k < num_series; ++k) {
          if (ord_times[k] == cur_time) {
            code |= 1 << k;

            /* Next spike for this neuron */
            ++(ord_iter[k]);

            if (ord_iter[k] == ord_end[k]) {
              ord_times[k] = end_time + 1;
            }
            else {
              ord_times[k] = (int)*(ord_iter[k]) - ord_shift[k];
            }
          }

          /* Find minimum next time bin */
          if (ord_times[k] < next_time) {
            next_time = ord_times[k];
          }
        }

        ++(counts[code]);
        cur_time = next_time;

      } /* while spikes left */

      /* Fill in zero count */
      counts[0] = end_time;
      for (k = 1; k < num_counts; ++k) {
        counts[0] -= counts[k];
      }

      /* ===================================================================== */

      /* Use counts to calculate TE */
      te_final = 0;

      /* Order is x^(k), y^(l), x(n+1) */
      for (k = 0; k < num_counts; ++k) {
        prob_1 = (double)counts[k] / (double)end_time;

        if (prob_1 == 0) {
          continue;
        }

        prob_2 = (double)counts[k] / (double)(counts[k] + counts[k ^ 1]);

        c1 = 0;
        c2 = 0;

        for (l = 0; l < num_y; ++l) {
          idx = (k & (num_x - 1)) + (l << (x_order + 1));
          c1 += counts[idx];
          c2 += (counts[idx] + counts[idx ^ 1]);
        }

        prob_3 = (double)c1 / (double)c2;

        te_final += (prob_1 * log2(prob_2 / prob_3));           
        slte_result[cell] += (prob_1 * log2(prob_2 / prob_3)) * rule[k];//SLTE

      }

      /* MATLAB is column major, but flipped for compatibility */

       te_result[cell] = te_final;


} /* transent_1 */
